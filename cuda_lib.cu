#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include<stdio.h>

__global__ void cuda_gray_kernel(unsigned char *b, unsigned char *g, unsigned char *r, unsigned char *gray, size_t size)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }
    
    gray[idx] = (unsigned char)(0.114f*b[idx] + 0.587f*g[idx] + 0.299f*r[idx] + 0.5);

    //printf("idx: %lu %uc %uc %uc %uc\n\n", idx, b[idx], g[idx], r[idx], gray[idx]);

    //gray[idx] = (int)(0.11*b[idx] + 0.59*g[idx] + 0.3*r[idx] + 0.5);
    //printf("%f\t%d\n\n", 0.11*b[idx] + 0.59*g[idx] + 0.3*r[idx], (int)(0.11*b[idx] + 0.59*g[idx] + 0.3*r[idx]));

}

extern "C" {
void cuda_gray(unsigned char *a, unsigned char *b, unsigned char *c, unsigned char *d, size_t size)
{

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    unsigned char *d_a, *d_b, *d_c, *d_d;

    hipMalloc((void **)&d_a, size * sizeof(char));
    hipMalloc((void **)&d_b, size * sizeof(char));
    hipMalloc((void **)&d_c, size * sizeof(char));
    hipMalloc((void **)&d_d, size * sizeof(char));

    hipMemcpy(d_a, a, size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, size * sizeof(char), hipMemcpyHostToDevice);


    hipEventRecord(start);
    cuda_gray_kernel <<< ceil(size / 1024.0), 1024 >>> (d_a, d_b, d_c, d_d, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time on GPU : %f msec\n", milliseconds);

    hipMemcpy(d, d_d, size * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
}
}
